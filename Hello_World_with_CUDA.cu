#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void outputFromGPU() {

    printf("Hello World from the GPU!\n");
}

int main(void) {

    printf("Hello Word from the CPU!\n");

    outputFromGPU<<<1,1>>>();

    hipDeviceSynchronize();

    printf("Hello World from the CPU AGAIN!\n");

    return 0;
}